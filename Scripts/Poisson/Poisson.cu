#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <fstream>
using namespace std;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		cerr<<"CUDA Runtime Error:"<<hipGetErrorString(result)<<endl;
		assert(result == hipSuccess);
	}
#endif
	return result;
}

bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if (count == 0) {
		cerr << "There is no device." << endl;
		return false;
	}

	int iX;
	for (iX = 0; iX < count; iX++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, iX) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (iX == count) {
		cerr << "There is no device supporting CUDA 1.x." << endl;
		return false;
	}

	hipSetDevice(iX);

	return true;
}

const int gXSize = 32, gYSize = 32, gZSize = 32;
const int gXSizeBlock = 8, gYSizeBlock = 8, gZSizeBlock = 16;
const int gSizeBlock = gXSize*gYSize*gZSize / (gXSizeBlock*gYSizeBlock*gZSizeBlock);

void CheckResults(double &error, double &maxError, double *result, double *reference,int length)
{
	maxError = 0.0;
	error = 0.0;
	for (int iNum = 0; iNum < length; ++iNum) {
		double o1Temp = result[iNum];
		double o2Temp = reference[iNum];
		error += (o1Temp - o2Temp)*(o1Temp - o2Temp);
		if (fabs(o1Temp - o2Temp) > maxError) maxError = fabs(o1Temp - o2Temp);
	}
	error = sqrt(error / (length));
}

__device__ unsigned int countBlock = 0;

__global__ void FDMCUDA(double *charge, double *potential,double interval)
{
	volatile __shared__ double sPotential[gZSizeBlock + 2][gYSizeBlock][gXSizeBlock];
	double omega = 1.0 / 6.0;
	double parameter = interval*interval/8.854187817e-12;
	int xThread = threadIdx.x;
	int yThread = threadIdx.y;
	int zThread = threadIdx.z;
	int zShared = threadIdx.z+1;
	int zSharedUp = blockDim.z + 1;
	int xGlobal = blockIdx.x*(blockDim.x - 2) + xThread;
	int yGlobal = blockIdx.y*(blockDim.y - 2) + yThread;
	int zGlobal = blockIdx.z*blockDim.z + zThread;
	int globalIdx = zGlobal * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	//int globalIdx = xGlobal * gYSize * gZSize + yGlobal * gZSize + zGlobal;
	int bottomIdx = (zGlobal + gZSize - 2) % (gZSize - 1)*gXSize * gYSize + yGlobal * gXSize + xGlobal;
	int topIdx = (zGlobal + blockDim.z) % (gZSize - 1)*gXSize * gYSize + yGlobal * gXSize + xGlobal;
	double chargeLocal = charge[globalIdx];
	int xRemainder = xThread % (blockDim.x - 1);
	int yRemainder = yThread % (blockDim.y - 1);
	int zRemainder = zThread % (blockDim.z - 1);
	bool isSection = zThread == 0 && xRemainder != 0 && yRemainder != 0;
	bool isSurface = (xThread % (blockDim.x - 3) == 1 || yThread % (blockDim.y - 3) == 1 || zRemainder == 0) && xRemainder != 0 && yRemainder != 0;
	bool isShell = (xRemainder == 0 && yRemainder != 0) || (yRemainder == 0 && xRemainder != 0);

	sPotential[zShared][yThread][xThread] = potential[globalIdx];
	if (isSection)
	{
		sPotential[0][yThread][xThread] = potential[bottomIdx];
		sPotential[zSharedUp][yThread][xThread] = potential[topIdx];
	}
	__syncthreads();
	
	double potentialLocal = sPotential[zShared][yThread][xThread];

	if (xRemainder*yRemainder!=0)
	{
		potentialLocal = (sPotential[zShared + 1][yThread][xThread] + sPotential[zShared - 1][yThread][xThread] + sPotential[zShared][yThread + 1][xThread] + sPotential[zShared][yThread - 1][xThread] + sPotential[zShared][yThread][xThread + 1] + sPotential[zShared][yThread][xThread - 1] - parameter*chargeLocal)/6.0;
	}
	__syncthreads();
	for (int i = 0; i < 1; ++i)
	{
		//potentialLocal = (sPotential[zShared + 1][yThread][xThread] + sPotential[zShared - 1][yThread][xThread] + sPotential[zShared][yThread + 1][xThread] + sPotential[zShared][yThread - 1][xThread] + sPotential[zShared][yThread][xThread + 1] + sPotential[zShared][yThread][xThread - 1] - parameter*chargeLocal) / 6.0;


		/*
		sPotential[zShared][yThread][xThread] = potentialLocal;
		if (isSurface)
		{
			potential[globalIdx] = potentialLocal;
		}
		__threadfence();
		if (isShell)
		{
			sPotential[zShared][yThread][xThread] = potential[globalIdx];
		}
		if (isSection)
		{
			sPotential[0][yThread][xThread] = potential[bottomIdx];
			sPotential[zSharedUp][yThread][xThread] = potential[topIdx];
		}
		__syncthreads();
		if (xRemainder*yRemainder != 0)
		{
			potentialLocal = (sPotential[zShared + 1][yThread][xThread] + sPotential[zShared - 1][yThread][xThread] + sPotential[zShared][yThread + 1][xThread] + sPotential[zShared][yThread - 1][xThread] + sPotential[zShared][yThread][xThread + 1] + sPotential[zShared][yThread][xThread - 1] - parameter*chargeLocal) / 6.0;
		}
		__syncthreads();

		*/
		/*if (zGlobal == 0 || zGlobal == 31){
			potentialLocal = 0;
		}*/
		
	}
	
	if (xRemainder*yRemainder != 0)
	{
		potential[globalIdx] = (sPotential[zShared - 1][yThread][xThread] - 3.06559e+014) / 6.0;
	}
	
}

__global__ void FDM2CUDA(double *charge, double *potential, double *potential2, double *error, double interval)
{
	__shared__ double sError[gZSizeBlock*gYSizeBlock*gXSizeBlock];
	double omega = 1.0 / 6.0;
	double parameter = interval*interval / 8.854187817e-12;
	int xThread = threadIdx.x;
	int yThread = threadIdx.y;
	int zThread = threadIdx.z;
	int idThread = zThread*blockDim.x*blockDim.y + yThread*blockDim.x + xThread;
	int xGlobal = blockIdx.x*blockDim.x + xThread;
	int yGlobal = blockIdx.y*blockDim.y + yThread;
	int zGlobal = blockIdx.z*blockDim.z + zThread;
	int blockSize = gridDim.x*gridDim.y*gridDim.z;
	int globalIdx = zGlobal * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	double chargeLocal = charge[globalIdx];
	int xFront = globalIdx - 1;
	int xNext = globalIdx + 1;
	int yFront = globalIdx - gXSize;
	int yNext = globalIdx + gXSize;;
	int zFront = (zGlobal + gZSize - 2) % (gZSize - 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	int zNext = (zGlobal % (gZSize - 1) + 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	bool notShell = (xGlobal != 0 && xGlobal != gXSize - 1 && yGlobal != 0 && yGlobal != gYSize - 1);

	double potentialLocal = 0.0;
	double potentialOld = 0.0;

	/*for (int i = 0; i < 1; ++i){
		if (notShell){
			potentialLocal = potential[xFront];
			potentialLocal += potential[xNext];
			potentialLocal += potential[yFront];
			potentialLocal += potential[yNext];
			potentialLocal += potential[zFront];
			potentialLocal += potential[zNext];
			potentialLocal = (potentialLocal - parameter*chargeLocal) / 6.0;
		}
		__syncthreads();
		if (isFirst){
			atomicInc(&countBlock, blockSize);
			while (countBlock != 0){}
		}
		__syncthreads();
		potential[globalIdx] = potentialLocal;
		__syncthreads();
		if (isFirst){
			atomicInc(&countBlock, blockSize);
			while (countBlock != 0){}
		}
		__syncthreads();
	}*/

	if (notShell){
		potentialOld = potential[globalIdx];
		potentialLocal = potentialOld + omega*(potential[xFront] + potential[xNext] + potential[yFront] + potential[yNext] + potential[zFront] + potential[zNext] - 6 * potentialOld - parameter*chargeLocal);
	}
	//sPotential[zThread][yThread][xThread] = potentialLocal;
	potential2[globalIdx] = potentialLocal;
	if (potentialLocal == 0.0){
		sError[idThread] = 0.0;
	}
	else{
		sError[idThread] = fabs((potentialLocal - potentialOld) / potentialLocal);
	}
	__syncthreads();
	for (unsigned int iNum = (blockDim.x * blockDim.y * blockDim.z) / 2; iNum > 0; iNum >>= 1){
		if (idThread < iNum){
			sError[idThread] += sError[idThread + iNum];
		}
		__syncthreads();
	}
	if (idThread == 0){
		error[blockIdx.z*gridDim.x*gridDim.y + blockIdx.y*gridDim.x + blockIdx.x] = sError[idThread] / (gZSizeBlock*gYSizeBlock*gXSizeBlock);
	}
}

__global__ void FDM3CUDA(double *charge, double *potential, double *error, double interval){
	__shared__ double sError[gZSizeBlock*gYSizeBlock*gXSizeBlock];
	double omega = 1.86/ 6.0;
	double parameter = interval*interval / 8.854187817e-12;
	int xThread = threadIdx.x;
	int yThread = threadIdx.y;
	int zThread = threadIdx.z;
	int idThread = zThread*blockDim.x*blockDim.y + yThread*blockDim.x + xThread;
	int xGlobal = blockIdx.x*blockDim.x + xThread;
	int yGlobal = blockIdx.y*blockDim.y + yThread;
	//int zGlobal = blockIdx.z*blockDim.z + zThread;
	int zGlobal = (blockIdx.z*blockDim.z + zThread) * 2 + (xGlobal + yGlobal)%2;
	int blockSize = gridDim.x*gridDim.y*gridDim.z;
	int globalIdx = zGlobal * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	double chargeLocal = -charge[globalIdx];
	int xFront = globalIdx - 1;
	int xNext = globalIdx + 1;
	int yFront = globalIdx - gXSize;
	int yNext = globalIdx + gXSize;;
	int zFront = (zGlobal + gZSize - 2) % (gZSize - 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	int zNext = (zGlobal % (gZSize - 1) + 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	bool notShell = (xGlobal != 0 && xGlobal != gXSize - 1 && yGlobal != 0 && yGlobal != gYSize - 1 && zGlobal != gZSize - 1);
	//bool notShell = (xGlobal != 0 && xGlobal != gXSize - 1 && yGlobal != 0 && yGlobal != gYSize - 1 && zGlobal != gZSize - 1 && zGlobal != 0);

	double potentialLocal = 0.0;
	double potentialOld = 0.0;

	if (notShell){
		potentialOld = potential[globalIdx];
		//potentialLocal = (potential[xFront] + potential[xNext] + potential[yFront] + potential[yNext] + potential[zFront] + potential[zNext] - parameter*chargeLocal) / 6.0;

		potentialLocal = potentialOld + omega*(potential[xFront] + potential[xNext] + potential[yFront] + potential[yNext] + potential[zFront] + potential[zNext] - 6*potentialOld - parameter*chargeLocal);
		potential[globalIdx] = potentialLocal;
	}
	if (zGlobal == 0){
		potential[(gZSize - 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal] = potentialLocal;
	}
	if (potentialLocal == 0.0){
		sError[idThread] = 0.0;
	}
	else{
		sError[idThread] = fabs((potentialLocal - potentialOld) / potentialLocal);
	}
	__syncthreads();
	for (unsigned int iNum = (blockDim.x * blockDim.y * blockDim.z) / 2; iNum > 0; iNum >>= 1){
		if (idThread < iNum){
			sError[idThread] += sError[idThread + iNum];
		}
		__syncthreads();
	}
	if (idThread == 0){
		error[blockIdx.z*gridDim.x*gridDim.y + blockIdx.y*gridDim.x + blockIdx.x] = sError[idThread] / (gZSizeBlock*gYSizeBlock*gXSizeBlock);
	}
}

__global__ void FDM4CUDA(double *charge, double *potential, double *error, double interval){
	__shared__ double sError[gZSizeBlock*gYSizeBlock*gXSizeBlock];
	double omega = 1.86 / 6.0;
	double parameter = interval*interval / 8.854187817e-12;
	int xThread = threadIdx.x;
	int yThread = threadIdx.y;
	int zThread = threadIdx.z;
	int idThread = zThread*blockDim.x*blockDim.y + yThread*blockDim.x + xThread;
	int xGlobal = blockIdx.x*blockDim.x + xThread;
	int yGlobal = blockIdx.y*blockDim.y + yThread;
	//int zGlobal = blockIdx.z*blockDim.z + zThread;
	int zGlobal = (blockIdx.z*blockDim.z + zThread) * 2 + (xGlobal + yGlobal+1) % 2;
	int blockSize = gridDim.x*gridDim.y*gridDim.z;
	int globalIdx = zGlobal * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	double chargeLocal = -charge[globalIdx];
	int xFront = globalIdx - 1;
	int xNext = globalIdx + 1;
	int yFront = globalIdx - gXSize;
	int yNext = globalIdx + gXSize;;
	int zFront = (zGlobal + gZSize - 2) % (gZSize - 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	int zNext = (zGlobal % (gZSize - 1) + 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal;
	bool notShell = (xGlobal != 0 && xGlobal != gXSize - 1 && yGlobal != 0 && yGlobal != gYSize - 1 && zGlobal != gZSize - 1);
	//bool notShell = (xGlobal != 0 && xGlobal != gXSize - 1 && yGlobal != 0 && yGlobal != gYSize - 1 && zGlobal != gZSize - 1 && zGlobal != 0);
	double potentialLocal = 0.0;
	double potentialOld = 0.0;

	if (notShell){
		potentialOld = potential[globalIdx];
		//potentialLocal = (potential[xFront] + potential[xNext] + potential[yFront] + potential[yNext] + potential[zFront] + potential[zNext] - parameter*chargeLocal) / 6.0;

		potentialLocal = potentialOld + omega*(potential[xFront] + potential[xNext] + potential[yFront] + potential[yNext] + potential[zFront] + potential[zNext] - 6 * potentialOld - parameter*chargeLocal);
		potential[globalIdx] = potentialLocal;
	}
	if (zGlobal == 0){
		potential[(gZSize - 1) * gXSize * gYSize + yGlobal * gXSize + xGlobal] = potentialLocal;
	}
	if (potentialLocal == 0.0){
		sError[idThread] = 0.0;
	}
	else{
		sError[idThread] = fabs((potentialLocal - potentialOld) / potentialLocal);
	}
	__syncthreads();
	for (unsigned int iNum = (blockDim.x * blockDim.y * blockDim.z) / 2; iNum > 0; iNum >>= 1){
		if (idThread < iNum){
			sError[idThread] += sError[idThread + iNum];
		}
		__syncthreads();
	}
	if (idThread == 0){
		error[blockIdx.z*gridDim.x*gridDim.y + blockIdx.y*gridDim.x + blockIdx.x] = sError[idThread] / (gZSizeBlock*gYSizeBlock*gXSizeBlock);
	}
}


__global__ void AvgCUDA(double *avg, double *array){
	__shared__ double sData[gSizeBlock/2];
	unsigned int idThread = threadIdx.x;
	sData[idThread] = array[idThread] + array[idThread + blockDim.x];
	for (unsigned int iNum = blockDim.x / 2; iNum > 0; iNum >>= 1){
		if (idThread < iNum){
			sData[idThread] += sData[idThread + iNum];
		}
		__syncthreads();
	}
	if (idThread == 0){
		*avg = sData[idThread] / (2 * blockDim.x);
	}
}

void InitInput(double *potential)
{
	int iPoten = 0;
	for (int iZ = 0; iZ < gZSize; ++iZ){
		for (int iY = 0; iY < gYSize; ++iY){
			for (int iX = 0; iX < gXSize; ++iX){
				if (iX*iY == 0 || iX == 31 || iY==31){
					potential[iZ*gXSize*gYSize + iY*gXSize + iX] = 0.0;
				}
				else{
					potential[iZ*gXSize*gYSize + iY*gXSize + iX] = 0.0;	
					//potential[iZ*gXSize*gYSize + iY*gXSize + iX] = (double)iPoten;
					++iPoten;
				}
			}
		}
	}
}

int main()
{
	if (!InitCUDA()) return 0;
	if (((gXSize - 2) % (gXSizeBlock - 2) != 0) || ((gYSize - 2) % (gYSizeBlock - 2) != 0) || (gZSize % gZSizeBlock != 0)) {
		cerr<<"Size error!"<<endl;
		exit(1);
	}

	double *oCharge = new double[gXSize*gYSize*gZSize];
	double *oPotential = new double[gXSize*gYSize*gZSize];
	double oInterval = 1.;

	fstream oFile("qGrid.txt", ios::in);
	if (oFile.fail()) {
		cout << "Error opening the input file!" << endl;
		exit(1);
	}
	int iNum = 0;
	while (!oFile.eof()&&iNum<32768) 
	{
		oFile >> oCharge[iNum];
		oFile >> oCharge[iNum];
		oFile >> oCharge[iNum];
		oFile >> oCharge[iNum];
		//oCharge[iNum] = (double)iNum;
	//	oCharge[iNum] = 0;
		++iNum;
	}
	oFile.close();

	//InitInput(oCharge);
	//oCharge[10 * gXSize*gYSize + 10 * gXSize + 10] = 100;
	InitInput(oPotential);

	int bytes = gXSize*gYSize*gZSize * sizeof(double);
	int iCount = 0;
	double oError=1;
	double *dCharge, *dPotential, *d2Potential, *dErrorBlock, *dError;
	checkCuda(hipMalloc((void**)&dCharge, bytes));
	checkCuda(hipMalloc((void**)&dPotential, bytes));
	checkCuda(hipMalloc((void**)&d2Potential, bytes));
	//checkCuda(hipMalloc((void**)&dErrorBlock, gSizeBlock*sizeof(double)));
	checkCuda(hipMalloc((void**)&dErrorBlock, gSizeBlock*sizeof(double)/2));
	checkCuda(hipMalloc((void**)&dError, sizeof(double)));
	float timeCUDA;
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));

	//double error, maxError;

	checkCuda(hipMemcpy(dCharge, oCharge, bytes, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(dPotential, oPotential, bytes, hipMemcpyHostToDevice));

	dim3 grid((gXSize-2) / (gXSizeBlock-2), (gYSize-2) / (gYSizeBlock-2), gZSize / gZSizeBlock);
	//dim3 grid2(gXSize / gXSizeBlock, gYSize / gYSizeBlock, gZSize / gZSizeBlock);
	dim3 grid2(gXSize / gXSizeBlock, gYSize / gYSizeBlock, gZSize / (2 * gZSizeBlock));
	dim3 block(gXSizeBlock, gYSizeBlock, gZSizeBlock);

	checkCuda(hipEventRecord(startEvent, 0));
	//FDMCUDA << <grid, block >> >(dCharge, dPotential,oInterval);
	while (oError > 1e-7&&iCount<5000){
		FDM3CUDA << <grid2, block >> >(dCharge, dPotential, dErrorBlock, oInterval);
		//FDM4CUDA << <grid2, block >> >(dCharge, d2Potential, dPotential, dErrorBlock, oInterval);
		FDM4CUDA << <grid2, block >> >(dCharge, dPotential, dErrorBlock, oInterval);
		AvgCUDA << <1, gSizeBlock / 2 >> >(dError, dErrorBlock);
		checkCuda(hipMemcpy(&oError, dError, sizeof(double), hipMemcpyDeviceToHost));
		++iCount;
		cout << iCount << ":\t" << oError << endl;
	}
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&timeCUDA, startEvent, stopEvent));

	checkCuda(hipMemcpy(oPotential, dPotential, bytes, hipMemcpyDeviceToHost));

	cout << "Count:" << iCount << endl;
	cout << "Time:" << timeCUDA << "ms" << endl;

	oFile.open("output.txt", ios::out);
	for (int iZ=0; iZ < gZSize; ++iZ){
		for (int iY=0; iY < gYSize; ++iY){
			for (int iX=0; iX < gXSize; ++iX){
				oFile << iX << "\t" << iY << "\t" << iZ << "\t" << oPotential[iZ*gXSize*gYSize + iY*gXSize + iX] << endl;
			}
		}
	}
	oFile.close();

	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));

	checkCuda(hipFree(dCharge));
	checkCuda(hipFree(dPotential));
	checkCuda(hipFree(dError));
	checkCuda(hipFree(dErrorBlock));

	delete[] oCharge;
	delete[] oPotential;

	system("pause");
}